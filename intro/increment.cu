// simple increment kernel

#include <hip/hip_runtime.h>
#include <stdio.h>

//TODO: increment kernel
__global__
void increment(float *val) {
    *val += 2;
}

int main(void)
{
    // create host array and initialize
    float* device_pointer;
    
    // allocate device memory    
    hipMalloc(&device_pointer, sizeof(float));
    
    // print original value
    float* host_pointer = (float*)malloc(sizeof(float));
    *host_pointer = 40;
    printf("%f\n", *host_pointer);

    // memcpy to device
    hipMemcpy(device_pointer, host_pointer, sizeof(float), hipMemcpyHostToDevice);

    // launch the increment kernel
    increment <<< 1, 1 >>> (device_pointer);

    // memcpy results back to host
    hipMemcpy(host_pointer, device_pointer, sizeof(float), hipMemcpyDeviceToHost);

    // print new value
    printf("%f\n", *host_pointer);

    hipFree(device_pointer);

    return 0;
}
