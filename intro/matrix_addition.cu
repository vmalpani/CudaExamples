
/*************************************************
** Accelereyes Training Day 1					**
** Matrix Addition								**
** 												**
** This program will add two matrices and store **
** the result in a third matrix using the GPU	**
*************************************************/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define THREADS 10
using namespace std;

__global__ void add(int *a, int *b, int *c,int columns,int rows)
{
	// get the global id for the thread

	// calculate the index of the input data

    // perform addition
}

int main(void)
{
	int rows = 100;
	int columns = 100;
	int elements = rows * columns;

	size_t size = rows * columns * sizeof(int);

	// create device pointers
	int* d_a;
	int* d_b;
	int* d_c;

	// allocate memory on the device
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size); hipMalloc(&d_c, size);

	// initalize host variables
	vector<int> h_a(elements, 5);
	vector<int> h_b(elements, 5);
	vector<int> h_c(elements);

	// transfer the host data to the GPU
	hipMemcpy(d_a, &h_a.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b.front(), size, hipMemcpyHostToDevice);

	// calculate the number of threads and blocks

	// Launch the add kernel

	// get the results from the GPU
	hipMemcpy(&h_c.front(), d_c, size, hipMemcpyDeviceToHost);

    // print top left corner
	for(int i = 0; i < 5; i++) {
		for(int j = 0; j < 10; j++)
			cout << h_c[i * rows + j] << " ";
		cout << endl;
	}
}

